#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <algorithm> 
#include <time.h>

#include "Inputs/A_10_2.h"
#include "Inputs/A_32.h"
#include "Inputs/A_512.h"
#include "Inputs/A_1024.h"

#include "Inputs/b_10.h"
#include "Inputs/b_32.h"
#include "Inputs/b_512.h"
#include "Inputs/b_1024.h"
#include "Inputs/X_32.h"
#include "Inputs/X_512.h"
#include "Inputs/X_1024.h"

// ------------------------------------------------
using namespace std;
#include <iostream>

__global__ void gpu_process(float* x_temp, float* d_a, float* d_b, int num_threads, int num_blocks);
__global__ void sum_temp(float* x_temp, float* result, int num_threads, int num_blocks);
void pre_process(float** x_temp, float** x, float** d_A, float** d_B, float* A, float* B);
float run_process(int num_threads, float* d_a, float* d_b, float* x_temp, float* x);

const int N = 10;

void displayVector(float A[N]);
void displayFlat(float A[N * N]);
bool inverse(float A[N][N], float* inverse);
void adjoint(float A[N][N], float adj[N][N]);
float determinant(float A[N][N], int n);
void getCofactor(float A[N][N], float temp[N][N], int p, int q, int n);
void subtract(float* output, float A[N], float B[N]);
void display(float A[N][N]);

int main()
{
	float* x_temp, * x, * d_A, * d_B;
	float* difference = (float*)malloc(N * sizeof(float));

	if (N == 10)
	{
		float* A_orig = (float*)malloc(N * N * sizeof(float));
		float* d_orig_A;
		float inv_A[N * N];
		int number_of_threads = 2048;
		float time_spent_inverse, time_spent_multiplication;

		clock_t begin = clock();
		inverse(A_10, inv_A);
		time_spent_inverse = (float)(clock() - begin) / CLOCKS_PER_SEC;

		cout << "The input matrix is: \n";
		display(A_10);
		cout << "\nThe inverse is: \n";
		displayFlat(inv_A);
		printf("\nTime Spent Inversing matrix: %d seconds \n", time_spent_inverse);

		size_t matrixAsize = N * N * sizeof(float);
		hipMalloc((void**) &d_orig_A, matrixAsize);

		// Flatten A
		for (int i = 0; i < N; i++) {
			for (int j = 0; j < N; j++) {
				// change argument here for different weight matrices
				A_orig[i * N + j] = A_10[i][j];
			}
		}

		hipMemcpy(d_orig_A, A_orig, matrixAsize, hipMemcpyHostToDevice);
		pre_process(&x_temp, &x, &d_A, &d_B, inv_A, b_10);

		begin = clock();
		// inv_A * b
		run_process(number_of_threads, d_A, d_B, x_temp, x);
		cout << "X is: \n";
		displayVector(x);
		// A * x, x now holds b 
		run_process(number_of_threads, d_orig_A, x, x_temp, x);
		time_spent_multiplication = (float)(clock() - begin) / CLOCKS_PER_SEC;

		subtract(difference, x, b_10);
		displayVector(difference);
	}
	else
	{
		//float* b_provided = b_512;
		//float* a_provided = A_512;
		//float* x_provided = X_512;

		//pre_process(&x_temp, &x, &d_A, &d_B, a_provided, x_provided);

		//if (N != 1024)
		//{
		//	// RUN FOR ALL OTHER SIZES WHERE TIMING INFO NOT NEEDED
		//	int number_of_threads = 2048;
		//	run_process(number_of_threads, d_A, d_B, x_temp, x);
		//	cout << "\nA*x is: \n";
		//	displayVector(x);
		//	subtract(difference, x, b_provided);
		//	cout << "\nA*x-B is: \n";
		//	displayVector(difference);
		//}
		//else
		//{
		//	// ONLY RUN FOR 1024X1024 matrix
		//	 //Run through matrix multiplication with numthreads 
		//	int max_thread_power = 11;
		//	printf("\nMatrix Dimension: %d \n", N);
		//	for (int i = 0; i <= max_thread_power; i++) {
		//		int number_of_threads = pow(2, i);
		//		float duration = run_process(number_of_threads, d_A, d_B, x_temp, x);
		//		cout << "\n Number of threads: " << number_of_threads << "\t Run time: " << scientific << duration;
		//		cout << "\nX is: \n";
		//		displayVector(x);
		//	}
		//}

	}
	free(difference);
	hipFree(x_temp);
	hipFree(x);
	hipFree(d_A);
	hipFree(d_B);

	return 0;
}

float run_process(int num_threads, float* d_a, float* d_b, float* x_temp, float* x)
{
	int block_number = pow(2, num_threads / 1024);
	int threads_per_block = num_threads / block_number;

	float time_spent = 0.0;
	clock_t begin = clock();

	gpu_process << <block_number, threads_per_block >> > (x_temp, d_a, d_b, threads_per_block, block_number);
	hipDeviceSynchronize();
	sum_temp << <block_number, threads_per_block >> > (x_temp, x, threads_per_block, block_number);
	//serial_sum_temp(x_temp, x);
	hipDeviceSynchronize();

	clock_t end = clock();
	time_spent += (float)(end - begin) / CLOCKS_PER_SEC;
	return time_spent;
}

// Intermediate step in matrix multiplication of d_a x d_b.
// Multiplications are done and stored into x_temp
__global__ void gpu_process(float* x_temp, float* d_a, float* d_b, int num_threads, int num_blocks)
{
	int start, end;

	int thread_size = (N * N) / (num_blocks * num_threads);
	if (thread_size == 0) thread_size = 1;

	start = thread_size * (blockIdx.x * blockDim.x + threadIdx.x);
	end = start + thread_size;

	if (end > (N * N))
	{
		end = (N * N);
	}

	for (int i = start; i < end; i++)
	{
		x_temp[i] = d_a[i] * d_b[i % N];
	}
}

__global__ void sum_temp(float* x_temp, float* result, int num_threads, int num_blocks)
{
	// reinit results
	for (int j = 0; j < N; j++)
	{
		result[j] = 0;
	}

	int start, end;
	int thread_size = N / (num_blocks * num_threads);
	if (thread_size == 0) thread_size = 1;

	start = thread_size * (blockIdx.x * blockDim.x + threadIdx.x);
	end = start + thread_size;

	if (end > N) {
		end = N;
	}

	for (int i = start; i < end; i++)
	{
		for (int j = 0; j < N; j++)
		{
			// sum up rows 
			result[i] += x_temp[i * N + j];
		}
	}
}

// Allocated gpu memory for A x B multiplication. A is matrix, B is vector
// d_A holds gpu version of A, d_B for B
// x_temp holds intermediate multiplications. Shared Mem. 
// x holds final vector result
void pre_process(float** x_temp, float** x, float** d_A, float** d_B, float* A, float* B)
{
	unsigned error;

	// allocate and copy into device
	size_t matrixAsize = (size_t)(N * N * sizeof(float));
	size_t matrixBsize = (size_t)(N * sizeof(float));

	hipMalloc((void**) & *d_A, matrixAsize);
	hipMalloc((void**) & *d_B, matrixBsize);
	hipMallocManaged((void**) & *x_temp, matrixAsize);

	hipMemcpy(*d_A, A, matrixAsize, hipMemcpyHostToDevice);
	hipMemcpy(*d_B, B, matrixBsize, hipMemcpyHostToDevice);

	// allocate shared memory for x
	hipMallocManaged(x, matrixBsize);
}

void subtract(float* output, float A[N], float B[N])
{
	for (int i = 0; i < N; i++) {
		output[i] = A[i] - B[i];
	}
}

bool inverse(float A[N][N], float* inverse)
{
	float det = determinant(A, N);
	if (det == 0)
	{
		cout << "No inverse";
		return false;
	}

	// Find adjoint 
	float adj[N][N];
	adjoint(A, adj);

	// Find Inverse using formula "inverse(A) = adj(A)/det(A)" 
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			inverse[i * N + j] = adj[i][j] / float(det);

	return true;
}

void getCofactor(float A[N][N], float temp[N][N], int p, int q, int n)
{
	int i = 0, j = 0;

	// Looping for each element of the matrix 
	for (int row = 0; row < n; row++)
	{
		for (int col = 0; col < n; col++)
		{
			//  Copying into temporary matrix only those element 
			//  which are not in given row and column 
			if (row != p && col != q)
			{
				temp[i][j++] = A[row][col];

				// Row is filled, so increase row index and 
				// reset col index 
				if (j == n - 1)
				{
					j = 0;
					i++;
				}
			}
		}
	}
}

float determinant(float A[N][N], int n)
{
	float D = 0; // Initialize result 

	//  Base case : if matrix contains single element 
	if (n == 1)
		return A[0][0];

	float temp[N][N]; // To store cofactors 

	int sign = 1;  // To store sign multiplier 

	 // Iterate for each element of first row 
	for (int f = 0; f < n; f++)
	{
		// Getting Cofactor of A[0][f] 
		getCofactor(A, temp, 0, f, n);
		D += sign * A[0][f] * determinant(temp, n - 1);

		// terms are to be added with alternate sign 
		sign = -sign;
	}

	return D;
}

void adjoint(float A[N][N], float adj[N][N])
{
	if (N == 1)
	{
		adj[0][0] = 1;
		return;
	}

	// temp is used to store cofactors of A[][] 
	int sign = 1;
	float temp[N][N];

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			// Get cofactor of A[i][j] 
			getCofactor(A, temp, i, j, N);

			// sign of adj[j][i] positive if sum of row 
			// and column indexes is even. 
			sign = ((i + j) % 2 == 0) ? 1 : -1;

			// Interchanging rows and columns to get the 
			// transpose of the cofactor matrix 
			adj[j][i] = (sign) * (determinant(temp, N - 1));
		}
	}
}

void displayFlat(float A[N * N])
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			cout << A[i * N + j] << " ";
		cout << endl;
	}
}

void displayVector(float A[N])
{
	cout.precision(17);
	for (int i = 0; i < N; i++)
	{
		cout << A[i] << fixed << endl;
	}
}

void serial_sum_temp(float* x_temp, float* result)
{
	// reinit results
	for (int j = 0; j < N; j++)
	{
		result[j] = 0;
	}

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			result[i] += x_temp[i * N + j];
		}
	}
}


void display(float A[N][N])
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			cout << A[i][j] << " ";
		cout << endl;
	}
}
