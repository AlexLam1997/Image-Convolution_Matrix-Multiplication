#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <algorithm> 
#include <time.h>

#include "Inputs/A_10_2.h"
#include "Inputs/A_32_2.h"
#include "Inputs/A_512_2.h"
#include "Inputs/A_1024_2.h"

#include "Inputs/b_10.h"
#include "Inputs/b_32.h"
#include "Inputs/b_512.h"
#include "Inputs/b_1024.h"

// ------------------------------------------------
//#include <bits/stdc++.h> 
#define N 32 

using namespace std;
#include <iostream>

void getCofactor(double A[N][N], double temp[N][N], int p, int q, int n)
{
    int i = 0, j = 0;

    // Looping for each element of the matrix 
    for (int row = 0; row < n; row++)
    {
        for (int col = 0; col < n; col++)
        {
            //  Copying into temporary matrix only those element 
            //  which are not in given row and column 
            if (row != p && col != q)
            {
                temp[i][j++] = A[row][col];

                // Row is filled, so increase row index and 
                // reset col index 
                if (j == n - 1)
                {
                    j = 0;
                    i++;
                }
            }
        }
    }
}

double determinant(double A[N][N], int n)
{
    double D = 0; // Initialize result 

    //  Base case : if matrix contains single element 
    if (n == 1)
        return A[0][0];

    double temp[N][N]; // To store cofactors 

    int sign = 1;  // To store sign multiplier 

     // Iterate for each element of first row 
    for (int f = 0; f < n; f++)
    {
        // Getting Cofactor of A[0][f] 
        getCofactor(A, temp, 0, f, n);
        D += sign * A[0][f] * determinant(temp, n - 1);

        // terms are to be added with alternate sign 
        sign = -sign;
    }

    return D;
}

void adjoint(double A[N][N], double adj[N][N])
{
    if (N == 1)
    {
        adj[0][0] = 1;
        return;
    }

    // temp is used to store cofactors of A[][] 
    int sign = 1;
    double temp[N][N];

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            // Get cofactor of A[i][j] 
            getCofactor(A, temp, i, j, N);

            // sign of adj[j][i] positive if sum of row 
            // and column indexes is even. 
            sign = ((i + j) % 2 == 0) ? 1 : -1;

            // Interchanging rows and columns to get the 
            // transpose of the cofactor matrix 
            adj[j][i] = (sign) * (determinant(temp, N - 1));
        }
    }
}

// Function to calculate and store inverse, returns false if 
// matrix is singular 
bool inverse(double A[N][N], double* inverse)
{
    // Find determinant of A[][] 
    double det = determinant(A, N);
    if (det == 0)
    {
        cout << "Singular matrix, can't find its inverse";
        return false;
    }

    // Find adjoint 
    double adj[N][N];
    adjoint(A, adj);

    // Find Inverse using formula "inverse(A) = adj(A)/det(A)" 
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            inverse[i*N +j] = adj[i][j] / double(det);

    return true;
}

void displayFlat(double A[N*N])
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            cout << A[i * N + j] << " ";
        cout << endl;
    }
}

void cudadisplayFlat(double A[N * N])
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            cout << A[i * N + j] << " ";
        cout << endl;
    }
}

void displayVector(double A[N])
{
    for (int i = 0; i < N; i++)
    {
        printf("%f ", A[i]);
    }
}

void display(double A[N][N])
{
    for (int i = 0; i < N*N; i++)
    {
        for (int j = 0; j < N; j++)
            cout << A[i][j] << " ";
        cout << endl;
    }
}

// -------------------------------------------------------------------------------------

__global__ void gpu_process(double* x_temp, double* d_a, double* d_b, int num_threads, int num_blocks)
{
    int start, end;
    
    int thread_size=(N*N)/(num_blocks * num_threads);    
    if (thread_size == 0) thread_size=1;

    start = thread_size* (blockIdx.x * blockDim.x + threadIdx.x);
    end = start+ thread_size;

    for (int i = start; i < end; i++)
    {
        x_temp[i]= d_b[i/N]*d_a[i];
    }
}

__global__ void sum_temp(double* x_temp, double* result, int num_threads, int num_blocks)
{
    int start, end;

    int thread_size = (N) / (num_blocks * num_threads);
    if (thread_size == 0) thread_size = 1;

    start = thread_size * (blockIdx.x * blockDim.x + threadIdx.x);
    end = start + thread_size;

    for (int i = start; i < end; i++)
    {
        for (int j = 0; j < N; j++)
        {
            result[i] += x_temp[i + j*N];
        }
    }
}

// void serial_sum_temp(double* x_temp, double* result)
//{
//
//    for (int i = 0; i < N; i++)
//    { 
//        for (int j = 0; j < N; j++)
//        {
//            result[i] += x_temp[j*N + i];
//        }
//    }
//}

double run_process(int num_threads, double* d_a, double* d_b, double* x_temp, double* x) {
    int block_number = num_threads / 1024 + 1;
    int threads_per_block = num_threads / block_number;

    double time_spent = 0.0;
    clock_t begin = clock();
    gpu_process << <block_number, threads_per_block >> > (x_temp, d_a, d_b, threads_per_block, block_number);
    hipDeviceSynchronize();
    
    cout << "\n X temp is: \n";
    displayFlat(x_temp);

    sum_temp<<<block_number, threads_per_block >>>(x_temp, x, threads_per_block, block_number);
    hipDeviceSynchronize();
    clock_t end = clock();

    time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
    return time_spent;
}

void pre_process(double** x_temp, double** x, double** d_A, double** d_B, double* A, double* B) {
    unsigned error;
    
    // allocate and copy into device
    size_t matrixAsize = (size_t) (N * N * sizeof(double));
    size_t matrixBsize = (size_t) (N * sizeof(double));

    hipMalloc((void**) & *d_A, matrixAsize);
    hipMalloc((void**) & *d_B, matrixBsize);
    hipMallocManaged((void**) & *x_temp, matrixAsize);

    hipMemcpy(*d_A, A, matrixAsize, hipMemcpyHostToDevice);
    hipMemcpy(*d_B, B, matrixBsize, hipMemcpyHostToDevice);

    // allocate shared memory for x
    hipMallocManaged(x, matrixBsize);
    
}


int main()
{
    double inv_A[N*N]; 
    
    int num_threads = 10;
    double* x_temp, * x, * d_A, * d_B;

    inverse(A_32, inv_A);
    cout << "The inverse is: \n";
    displayFlat(inv_A);

    pre_process(&x_temp, &x, &d_A, &d_B, inv_A, b_32);
    
    run_process(num_threads, d_A, d_B, x_temp, x);
    
    cout << "\n X is: \n";
    displayVector(x);
    return 0;
}

